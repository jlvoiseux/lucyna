#include "hip/hip_runtime.h"
#include "lyTensorMath.h"

#include <hip/hip_bf16.h>
#include <hip/hip_math_constants.h>
#include <stdio.h>

static int lyTensorGetElementCount(lyTensor* pTensor)
{
	int count = 1;
	for (int i = 0; i < pTensor->rank; i++)
	{
		count *= pTensor->shape[i];
	}
	return count;
}

static int32_t getTotalSize(const int32_t* shape, int32_t rank)
{
	int32_t size = 1;
	for (int32_t i = 0; i < rank; i++)
	{
		size *= shape[i];
	}
	return size;
}

__global__ void tensorMatMulKernel(hip_bfloat16* output, const hip_bfloat16* a, const hip_bfloat16* b, const int32_t* aStrides, const int32_t* bStrides, const int32_t* outStrides, const int32_t* aShape, const int32_t* bShape, int32_t rank, int32_t batchSize, int32_t m, int32_t n, int32_t k)
{
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batchSize * m * n)
		return;

	int32_t batchIdx = idx / (m * n);
	int32_t row		 = (idx % (m * n)) / n;
	int32_t col		 = idx % n;

	int32_t aOffset = 0;
	int32_t bOffset = 0;
	for (int32_t i = 0; i < rank - 2; i++)
	{
		int32_t dim = batchIdx / aStrides[i];
		aOffset += dim * aShape[i];
		bOffset += dim * bShape[i];
		batchIdx %= aStrides[i];
	}

	hip_bfloat16 sum = __float2bfloat16(0.0f);
	for (int32_t i = 0; i < k; i++)
	{
		int32_t aIdx = aOffset + row * k + i;
		int32_t bIdx = bOffset + i * n + col;
		sum			 = __hadd(sum, __hmul(a[aIdx], b[bIdx]));
	}
	output[idx] = sum;
}

__global__ void tensorScaleAndAddBroadcastKernel(hip_bfloat16* output, const hip_bfloat16* a, const hip_bfloat16* b, float alpha, float beta, int32_t* aShape, int32_t* bShape, int32_t aRank, int32_t bRank, int32_t totalElements)
{
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalElements)
		return;

	// Calculate indices for tensor A
	int32_t remaining = idx;
	int32_t aIndex	  = 0;
	int32_t bIndex	  = 0;
	int32_t stride	  = 1;

	// Calculate strided indices for both tensors
	for (int32_t i = aRank - 1; i >= 0; i--)
	{
		int32_t dim = remaining % aShape[i];
		remaining /= aShape[i];

		// For B, only use the last bRank dimensions
		if (i >= aRank - bRank)
		{
			int32_t bDim = dim % bShape[i - (aRank - bRank)];
			bIndex += bDim * stride;
		}

		aIndex += dim * stride;
		stride *= aShape[i];
	}

	hip_bfloat16 valA = __hmul(a[aIndex], __float2bfloat16(alpha));
	hip_bfloat16 valB = __hmul(b[bIndex], __float2bfloat16(beta));
	output[idx]		 = __hadd(valA, valB);
}

__global__ void tensorElementwiseMulKernel(hip_bfloat16* output, const hip_bfloat16* a, const hip_bfloat16* b, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

	output[idx] = __hmul(a[idx], b[idx]);
}

__global__ void triangularMaskKernel(hip_bfloat16* output, int32_t rows, int32_t cols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < rows && col < cols)
	{
		float val				 = col <= row ? 0.0f : -HIP_INF_F;
		output[row * cols + col] = __float2bfloat16(val);
	}
}

__global__ void tensorArgmaxKernel(hip_bfloat16* output, const hip_bfloat16* input, int32_t batchSize, int32_t dimSize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batchSize)
	{
		return;
	}

	float	maxVal = -HIP_INF_F;
	int32_t maxIdx = 0;

	for (int32_t i = 0; i < dimSize; i++)
	{
		float val = __bfloat162float(input[idx * dimSize + i]);
		if (val > maxVal)
		{
			maxVal = val;
			maxIdx = i;
		}
	}

	output[idx] = __float2bfloat16((float)maxIdx);
}

__global__ void tensorOuterKernel(hip_bfloat16* output, const hip_bfloat16* a, const hip_bfloat16* b, int aSize, int bSize)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < aSize && col < bSize)
	{
		output[row * bSize + col] = __hmul(a[row], b[col]);
	}
}

__global__ void tensorEmbeddingKernel(hip_bfloat16* output, const hip_bfloat16* tokens, const hip_bfloat16* embeddings, int seqLen, int dim)
{
	int idx	   = blockIdx.x * blockDim.x + threadIdx.x;
	int dimPos = idx % dim;
	int seqPos = idx / dim;

	if (seqPos >= seqLen)
		return;

	hip_bfloat16 tokenValue = tokens[seqPos];
	int			tokenId	   = (int)__bfloat162float(tokenValue);

	if (tokenId < 0)
	{
		output[idx] = __float2bfloat16(0.0f);
		return;
	}

	// Load embedding value
	hip_bfloat16 embedValue = embeddings[tokenId * dim + dimPos];
	output[idx]			   = embedValue;
}

__global__ void tensorTransposeKernel2(hip_bfloat16* output, const hip_bfloat16* input, const int32_t* dims, const int32_t* axesMap, int32_t rank, size_t totalElements)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= totalElements)
		return;

	int32_t row	   = idx / dims[1];
	int32_t col	   = idx % dims[1];
	size_t	dstIdx = col * dims[0] + row;
	output[dstIdx] = input[idx];
}

__device__ size_t getLinearIndex(const int32_t* indices, const int32_t* dims, int32_t rank)
{
	size_t linearIdx = 0;
	size_t stride	 = 1;

	for (int32_t i = rank - 1; i >= 0; i--)
	{
		linearIdx += indices[i] * stride;
		stride *= dims[i];
	}
	return linearIdx;
}

__device__ void getIndices(size_t linearIdx, int32_t* indices, const int32_t* dims, int32_t rank)
{
	size_t remaining = linearIdx;

	for (int32_t i = rank - 1; i >= 0; i--)
	{
		indices[i] = remaining % dims[i];
		remaining /= dims[i];
	}
}

__global__ void tensorTransposeKernel3(hip_bfloat16* output, const hip_bfloat16* input, const int32_t* dims, const int32_t* perm, int32_t rank, size_t totalElements)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= totalElements)
		return;

	// Calculate strides for input and output
	int32_t input_strides[8];
	int32_t output_strides[8];

	input_strides[rank - 1]	 = 1;
	output_strides[rank - 1] = 1;

	for (int i = rank - 2; i >= 0; i--)
	{
		input_strides[i]  = input_strides[i + 1] * dims[i + 1];
		output_strides[i] = output_strides[i + 1] * dims[perm[i + 1]];
	}

	// Calculate input indices
	int32_t input_idx  = idx;
	int32_t output_idx = 0;

	for (int i = 0; i < rank; i++)
	{
		int32_t dim_idx = input_idx / input_strides[i];
		input_idx		= input_idx % input_strides[i];
		output_idx += dim_idx * output_strides[perm[i]];
	}

	output[output_idx] = input[idx];
}

bool lyTensorMatMul(lyTensor** ppOutput, lyTensor* pA, lyTensor* pB)
{
	if (pA->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pA);
	}

	if (pB->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pB);
	}

	if (!ppOutput || !pA || !pB || pA->rank < 2 || pB->rank < 2)
		return false;

	if (pA->rank != pB->rank)
		return false;

	for (int32_t i = 0; i < pA->rank - 2; i++)
	{
		if (pA->shape[i] != pB->shape[i])
			return false;
	}

	int32_t m = pA->shape[pA->rank - 2];
	int32_t k = pA->shape[pA->rank - 1];
	int32_t n = pB->shape[pB->rank - 1];
	if (k != pB->shape[pB->rank - 2])
		return false;

	int32_t batchSize = 1;
	for (int32_t i = 0; i < pA->rank - 2; i++)
	{
		batchSize *= pA->shape[i];
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput, LY_MEMORY_GPU))
		return false;

	int32_t* outShape = (int32_t*)malloc(sizeof(int32_t) * pA->rank);
	if (!outShape)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	for (int32_t i = 0; i < pA->rank - 2; i++)
	{
		outShape[i] = pA->shape[i];
	}
	outShape[pA->rank - 2] = m;
	outShape[pA->rank - 1] = n;

	if (!lySetTensorShape(pOutput, outShape, pA->rank) || !lySetTensorData(pOutput, NULL, getTotalSize(outShape, pA->rank) * sizeof(hip_bfloat16)))
	{
		free(outShape);
		lyDestroyTensor(pOutput);
		return false;
	}
	free(outShape);

	int32_t* aStrides	= (int32_t*)malloc(sizeof(int32_t) * pA->rank);
	int32_t* bStrides	= (int32_t*)malloc(sizeof(int32_t) * pB->rank);
	int32_t* outStrides = (int32_t*)malloc(sizeof(int32_t) * pA->rank);
	if (!aStrides || !bStrides || !outStrides)
	{
		free(aStrides);
		free(bStrides);
		free(outStrides);
		lyDestroyTensor(pOutput);
		return false;
	}

	int32_t *dAShape, *dBShape, *dAStrides, *dBStrides, *dOutStrides;
	hipMalloc(&dAShape, sizeof(int32_t) * pA->rank);
	hipMalloc(&dBShape, sizeof(int32_t) * pB->rank);
	hipMalloc(&dAStrides, sizeof(int32_t) * pA->rank);
	hipMalloc(&dBStrides, sizeof(int32_t) * pB->rank);
	hipMalloc(&dOutStrides, sizeof(int32_t) * pA->rank);

	hipMemcpy(dAShape, pA->shape, sizeof(int32_t) * pA->rank, hipMemcpyHostToDevice);
	hipMemcpy(dBShape, pB->shape, sizeof(int32_t) * pB->rank, hipMemcpyHostToDevice);
	hipMemcpy(dAStrides, aStrides, sizeof(int32_t) * pA->rank, hipMemcpyHostToDevice);
	hipMemcpy(dBStrides, bStrides, sizeof(int32_t) * pB->rank, hipMemcpyHostToDevice);
	hipMemcpy(dOutStrides, outStrides, sizeof(int32_t) * pA->rank, hipMemcpyHostToDevice);

	int32_t totalElements = batchSize * m * n;
	int32_t blockSize	  = 256;
	int32_t gridSize	  = (totalElements + blockSize - 1) / blockSize;

	hipDeviceSynchronize();
	tensorMatMulKernel<<<gridSize, blockSize>>>(pOutput->data, pA->data, pB->data, dAStrides, dBStrides, dOutStrides, dAShape, dBShape, pA->rank, batchSize, m, n, k);

	hipFree(dAShape);
	hipFree(dBShape);
	hipFree(dAStrides);
	hipFree(dBStrides);
	hipFree(dOutStrides);
	free(aStrides);
	free(bStrides);
	free(outStrides);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	lyTensorMoveToCPU(pA);
	lyTensorMoveToCPU(pB);
	lyTensorMoveToCPU(pOutput);

	*ppOutput = pOutput;
	return true;
}

bool lyTensorScaleAndAdd(lyTensor** ppOutput, lyTensor* pA, lyTensor* pB, float alpha, float beta)
{
	if (pA->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pA);
	}

	if (pB->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pB);
	}

	if (!ppOutput || !pA || !pB || !pA->data || !pB->data || pA->rank < 2 || pB->rank < 2)
		return false;

	// Verify shape compatibility for broadcasting
	if (pB->rank > pA->rank)
		return false;

	// Check if dimensions match for broadcasting
	for (int32_t i = 0; i < pB->rank; i++)
	{
		int32_t aIdx = pA->rank - pB->rank + i;
		if (pB->shape[i] != pA->shape[aIdx])
			return false;
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput, LY_MEMORY_GPU))
		return false;

	if (!lySetTensorShape(pOutput, pA->shape, pA->rank))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	int32_t totalElements = 1;
	for (int32_t i = 0; i < pA->rank; i++)
		totalElements *= pA->shape[i];

	if (!lySetTensorData(pOutput, NULL, totalElements * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	// Allocate and copy shape arrays to device
	int32_t *dAShape, *dBShape;
	hipMalloc(&dAShape, pA->rank * sizeof(int32_t));
	hipMalloc(&dBShape, pB->rank * sizeof(int32_t));
	hipMemcpy(dAShape, pA->shape, pA->rank * sizeof(int32_t), hipMemcpyHostToDevice);
	hipMemcpy(dBShape, pB->shape, pB->rank * sizeof(int32_t), hipMemcpyHostToDevice);

	int32_t blockSize = 256;
	int32_t numBlocks = (totalElements + blockSize - 1) / blockSize;

	hipDeviceSynchronize();
	tensorScaleAndAddBroadcastKernel<<<numBlocks, blockSize>>>(pOutput->data, pA->data, pB->data, alpha, beta, dAShape, dBShape, pA->rank, pB->rank, totalElements);

	hipFree(dAShape);
	hipFree(dBShape);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	lyTensorMoveToCPU(pA);
	lyTensorMoveToCPU(pB);
	lyTensorMoveToCPU(pOutput);

	*ppOutput = pOutput;
	return true;
}

bool lyTensorElementwiseMul(lyTensor** ppOutput, lyTensor* pA, lyTensor* pB)
{
	if (pA->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pA);
	}

	if (pB->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pB);
	}

	if (!ppOutput || !pA || !pB || !pA->data || !pB->data)
	{
		return false;
	}

	if (pA->rank != pB->rank)
	{
		return false;
	}
	for (int i = 0; i < pA->rank; i++)
	{
		if (pA->shape[i] != pB->shape[i])
		{
			return false;
		}
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput, LY_MEMORY_GPU))
	{
		return false;
	}

	if (!lySetTensorShape(pOutput, pA->shape, pA->rank) || !lySetTensorData(pOutput, NULL, lyTensorGetElementCount(pOutput) * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	int totalElements = 1;
	for (int i = 0; i < pA->rank; i++)
	{
		totalElements *= pA->shape[i];
	}

	int blockSize = 256;
	int gridSize  = (totalElements + blockSize - 1) / blockSize;

	hipDeviceSynchronize();
	tensorElementwiseMulKernel<<<gridSize, blockSize>>>(pOutput->data, pA->data, pB->data, totalElements);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	lyTensorMoveToCPU(pA);
	lyTensorMoveToCPU(pB);
	lyTensorMoveToCPU(pOutput);

	*ppOutput = pOutput;
	return true;
}

bool lyTensorMakeTriangularMask(lyTensor* pTensor)
{
	if (pTensor->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pTensor);
	}

	if (!pTensor || !pTensor->data || pTensor->rank != 2)
	{
		return false;
	}

	int32_t rows = pTensor->shape[0];
	int32_t cols = pTensor->shape[1];

	dim3 blockSize(16, 16);
	dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

	hipDeviceSynchronize();
	triangularMaskKernel<<<gridSize, blockSize>>>(pTensor->data, rows, cols);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		return false;
	}

	lyTensorMoveToCPU(pTensor);

	return true;
}

bool lyTensorArgmax(lyTensor** ppOutput, lyTensor* pInput, int32_t dim)
{
	if (pInput->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pInput);
	}

	if (!ppOutput || !pInput || dim < 0 || dim >= pInput->rank)
	{
		return false;
	}

	int32_t* newShape = (int32_t*)malloc(sizeof(int32_t) * (pInput->rank - 1));
	if (!newShape)
	{
		return false;
	}

	int32_t batchSize = 1;
	int32_t j		  = 0;
	for (int32_t i = 0; i < pInput->rank; i++)
	{
		if (i != dim)
		{
			newShape[j++] = pInput->shape[i];
			batchSize *= pInput->shape[i];
		}
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput, LY_MEMORY_GPU))
	{
		free(newShape);
		return false;
	}

	if (!lySetTensorShape(pOutput, newShape, pInput->rank - 1) || !lySetTensorData(pOutput, NULL, lyTensorGetElementCount(pOutput) * sizeof(hip_bfloat16)))
	{
		free(newShape);
		lyDestroyTensor(pOutput);
		return false;
	}
	free(newShape);

	int32_t dimSize	  = pInput->shape[dim];
	int32_t blockSize = 256;
	int32_t numBlocks = (batchSize + blockSize - 1) / blockSize;

	hipDeviceSynchronize();
	tensorArgmaxKernel<<<numBlocks, blockSize>>>(pOutput->data, pInput->data, batchSize, dimSize);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	lyTensorMoveToCPU(pInput);
	lyTensorMoveToCPU(pOutput);

	*ppOutput = pOutput;
	return true;
}

bool lyTensorOuter(lyTensor** ppOutput, lyTensor* pA, lyTensor* pB)
{
	if (pA->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pA);
	}

	if (pB->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pB);
	}

	if (!ppOutput || !pA || !pB || !pA->data || !pB->data)
	{
		return false;
	}

	if (pA->rank != 1 || pB->rank != 1)
	{
		return false;
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput, LY_MEMORY_GPU))
	{
		return false;
	}

	int32_t outputShape[] = {pA->shape[0], pB->shape[0]};
	if (!lySetTensorShape(pOutput, outputShape, 2) || !lySetTensorData(pOutput, NULL, lyTensorGetElementCount(pOutput) * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	dim3 blockSize(16, 16);
	dim3 gridSize((pB->shape[0] + blockSize.x - 1) / blockSize.x, (pA->shape[0] + blockSize.y - 1) / blockSize.y);

	hipDeviceSynchronize();
	tensorOuterKernel<<<gridSize, blockSize>>>(pOutput->data, pA->data, pB->data, pA->shape[0], pB->shape[0]);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	lyTensorMoveToCPU(pA);
	lyTensorMoveToCPU(pB);
	lyTensorMoveToCPU(pOutput);

	*ppOutput = pOutput;
	return true;
}

bool lyTensorEmbedding(lyTensor** ppOutput, lyTensor* pTokens, lyTensor* pEmbeddings)
{
	if (pTokens->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pTokens);
	}

	if (pEmbeddings->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pEmbeddings);
	}

	if (!ppOutput || !pTokens || !pEmbeddings || pTokens->rank != 1 || pEmbeddings->rank != 2)
	{
		return false;
	}

	// Add dimension checks
	int vocabSize = pEmbeddings->shape[0];
	int seqLen	  = pTokens->shape[0];
	int dim		  = pEmbeddings->shape[1];

	printf("Embedding dims: vocab=%d seqLen=%d dim=%d\n", vocabSize, seqLen, dim);

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput, LY_MEMORY_GPU))
	{
		return false;
	}

	int32_t outputShape[] = {seqLen, dim};
	if (!lySetTensorShape(pOutput, outputShape, 2) || !lySetTensorData(pOutput, NULL, seqLen * dim * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	// Adjust block size to match warp size
	int blockSize	  = 256;
	int totalElements = seqLen * dim;

	// Ensure grid size is aligned to warp size
	int numBlocks = (totalElements + blockSize - 1) / blockSize;
	numBlocks	  = ((numBlocks + 31) / 32) * 32;  // Align to warp size

	dim3 grid(numBlocks);
	dim3 block(blockSize);

	hipDeviceSynchronize();
	tensorEmbeddingKernel<<<grid, block>>>(pOutput->data, pTokens->data, pEmbeddings->data, seqLen, dim);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error: %s\n", hipGetErrorString(error));
		lyDestroyTensor(pOutput);
		return false;
	}

	lyTensorMoveToCPU(pTokens);
	lyTensorMoveToCPU(pEmbeddings);
	lyTensorMoveToCPU(pOutput);

	*ppOutput = pOutput;
	return true;
}

bool lyTensorTranspose(lyTensor** ppOutput, lyTensor* pInput, int32_t* pPerm)
{
	if (pInput->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pInput);
	}

	if (!ppOutput || !pInput || !pPerm || pInput->rank < 2)
	{
		return false;
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput, LY_MEMORY_GPU))
	{
		return false;
	}

	int32_t* newShape = (int32_t*)malloc(sizeof(int32_t) * pInput->rank);
	if (!newShape)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	for (int32_t i = 0; i < pInput->rank; i++)
	{
		newShape[i] = pInput->shape[pPerm[i]];
	}

	if (!lySetTensorShape(pOutput, newShape, pInput->rank))
	{
		free(newShape);
		lyDestroyTensor(pOutput);
		return false;
	}

	size_t totalElements = 1;
	for (int32_t i = 0; i < pInput->rank; i++)
	{
		totalElements *= pInput->shape[i];
	}

	if (!lySetTensorData(pOutput, NULL, totalElements * sizeof(hip_bfloat16)))
	{
		free(newShape);
		lyDestroyTensor(pOutput);
		return false;
	}

	int32_t *d_dims, *d_perm;
	if (hipMalloc(&d_dims, pInput->rank * sizeof(int32_t)) != hipSuccess || hipMalloc(&d_perm, pInput->rank * sizeof(int32_t)) != hipSuccess)
	{
		free(newShape);
		lyDestroyTensor(pOutput);
		return false;
	}

	if (hipMemcpy(d_dims, pInput->shape, pInput->rank * sizeof(int32_t), hipMemcpyHostToDevice) != hipSuccess || hipMemcpy(d_perm, pPerm, pInput->rank * sizeof(int32_t), hipMemcpyHostToDevice) != hipSuccess)
	{
		hipFree(d_dims);
		hipFree(d_perm);
		free(newShape);
		lyDestroyTensor(pOutput);
		return false;
	}

	int32_t blockSize = 256;
	int32_t numBlocks = (totalElements + blockSize - 1) / blockSize;

	hipDeviceSynchronize();
	if (pInput->rank == 2)
	{
		tensorTransposeKernel2<<<numBlocks, blockSize>>>(pOutput->data, pInput->data, d_dims, d_perm, pInput->rank, totalElements);
	}
	else
	{
		tensorTransposeKernel3<<<numBlocks, blockSize>>>(pOutput->data, pInput->data, d_dims, d_perm, pInput->rank, totalElements);
	}

	hipError_t error = hipGetLastError();

	hipFree(d_dims);
	hipFree(d_perm);
	free(newShape);

	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	lyTensorMoveToCPU(pInput);
	lyTensorMoveToCPU(pOutput);

	*ppOutput = pOutput;
	return true;
}