#include "hip/hip_runtime.h"
#include "lyTensorMath.h"

#include <hip/hip_bf16.h>

#define LY_MAX_RANK 4

static int lyTensorGetElementCount(const lyTensor* pTensor)
{
	int count = 1;
	for (int i = 0; i < pTensor->rank; i++)
	{
		count *= pTensor->shape[i];
	}
	return count;
}

static int32_t getTotalSize(const int32_t* shape, int32_t rank)
{
	int32_t size = 1;
	for (int32_t i = 0; i < rank; i++)
	{
		size *= shape[i];
	}
	return size;
}

void calculateStrides(int32_t* strides, const int32_t* shape, int32_t rank)
{
	strides[rank - 1] = 1;
	for (int32_t i = rank - 2; i >= 0; i--)
	{
		strides[i] = strides[i + 1] * shape[i + 1];
	}
}

__global__ void tensorAddKernel(hip_bfloat16* output, const hip_bfloat16* a, const hip_bfloat16* b, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

	output[idx] = __hadd(a[idx], b[idx]);
}

__global__ void tensorMatMulKernel(hip_bfloat16* output, const hip_bfloat16* a, const hip_bfloat16* b, const int32_t* aStrides, const int32_t* bStrides, const int32_t* outStrides, const int32_t* aShape, const int32_t* bShape, int32_t rank, int32_t batchSize, int32_t m, int32_t n, int32_t k)
{
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batchSize * m * n)
		return;

	int32_t batchIdx = idx / (m * n);
	int32_t row		 = (idx % (m * n)) / n;
	int32_t col		 = idx % n;

	int32_t aOffset = 0;
	int32_t bOffset = 0;
	for (int32_t i = 0; i < rank - 2; i++)
	{
		int32_t dim = batchIdx / aStrides[i];
		aOffset += dim * aShape[i];
		bOffset += dim * bShape[i];
		batchIdx %= aStrides[i];
	}

	hip_bfloat16 sum = __float2bfloat16(0.0f);
	for (int32_t i = 0; i < k; i++)
	{
		int32_t aIdx = aOffset + row * k + i;
		int32_t bIdx = bOffset + i * n + col;
		sum			 = __hadd(sum, __hmul(a[aIdx], b[bIdx]));
	}
	output[idx] = sum;
}

__global__ void tensorScaleAndAddKernel(hip_bfloat16* output, const hip_bfloat16* input, const hip_bfloat16* mask, float scale, int numRows, int numCols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < numRows && col < numCols)
	{
		int idx		= row * numCols + col;
		output[idx] = __hmul(input[idx], __float2bfloat16(scale));
		if (mask)
		{
			output[idx] = __hadd(output[idx], mask[idx]);
		}
	}
}

__global__ void tensorElementwiseMulKernel(hip_bfloat16* output, const hip_bfloat16* a, const hip_bfloat16* b, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

	output[idx] = __hmul(a[idx], b[idx]);
}

__global__ void triangularMaskKernel(hip_bfloat16* output, int32_t rows, int32_t cols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < rows && col < cols)
	{
		float val				 = col <= row ? 0.0f : -INFINITY;
		output[row * cols + col] = __float2bfloat16(val);
	}
}

__global__ void tensorArgmaxKernel(hip_bfloat16* output, const hip_bfloat16* input, int32_t batchSize, int32_t dimSize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batchSize)
	{
		return;
	}

	float	maxVal = -INFINITY;
	int32_t maxIdx = 0;

	for (int32_t i = 0; i < dimSize; i++)
	{
		float val = __bfloat162float(input[idx * dimSize + i]);
		if (val > maxVal)
		{
			maxVal = val;
			maxIdx = i;
		}
	}

	output[idx] = __float2bfloat16((float)maxIdx);
}

__global__ void tensorOuterKernel(hip_bfloat16* output, const hip_bfloat16* a, const hip_bfloat16* b, int aSize, int bSize)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < aSize && col < bSize)
	{
		output[row * bSize + col] = __hmul(a[row], b[col]);
	}
}

__global__ void tensorEmbeddingKernel(hip_bfloat16* output, const hip_bfloat16* tokens, const hip_bfloat16* embeddings, int seqLen, int dim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= seqLen * dim)
		return;

	int seqPos = idx / dim;
	int dimPos = idx % dim;

	int tokenId = (int)__bfloat162float(tokens[seqPos]);
	output[idx] = embeddings[tokenId * dim + dimPos];
}

__global__ void tensorTransposeKernel(hip_bfloat16* output, const hip_bfloat16* input, const int32_t* inputShape, const int32_t* inputStrides, const int32_t* outputStrides, const int32_t* perm, int32_t rank, int32_t elementCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= elementCount)
		return;

	int32_t inputIdx				 = idx;
	int32_t multiDimIdx[LY_MAX_RANK] = {0};

	for (int32_t i = rank - 1; i >= 0; i--)
	{
		multiDimIdx[i] = inputIdx / inputStrides[i];
		inputIdx %= inputStrides[i];
	}

	int32_t outputIdx = 0;
	for (int32_t i = 0; i < rank; i++)
	{
		outputIdx += multiDimIdx[perm[i]] * outputStrides[i];
	}

	output[outputIdx] = input[idx];
}

bool lyTensorAdd(lyTensor** ppOutput, const lyTensor* pA, const lyTensor* pB)
{
	if (!ppOutput || !pA || !pB || !pA->data || !pB->data)
	{
		return false;
	}

	if (pA->rank != pB->rank)
	{
		return false;
	}
	for (int i = 0; i < pA->rank; i++)
	{
		if (pA->shape[i] != pB->shape[i])
		{
			return false;
		}
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput))
	{
		return false;
	}

	if (!lySetTensorShape(pOutput, pA->shape, pA->rank) || !lySetTensorData(pOutput, NULL, lyTensorGetElementCount(pOutput) * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	int size	  = lyTensorGetElementCount(pOutput);
	int blockSize = 256;
	int gridSize  = (size + blockSize - 1) / blockSize;

	tensorAddKernel<<<gridSize, blockSize>>>(pOutput->data, pA->data, pB->data, size);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	*ppOutput = pOutput;
	return true;
}

bool lyTensorMatMul(lyTensor** ppOutput, const lyTensor* pA, const lyTensor* pB)
{
	if (!ppOutput || !pA || !pB || pA->rank < 2 || pB->rank < 2)
		return false;

	if (pA->rank != pB->rank)
		return false;

	for (int32_t i = 0; i < pA->rank - 2; i++)
	{
		if (pA->shape[i] != pB->shape[i])
			return false;
	}

	int32_t m = pA->shape[pA->rank - 2];
	int32_t k = pA->shape[pA->rank - 1];
	int32_t n = pB->shape[pB->rank - 1];
	if (k != pB->shape[pB->rank - 2])
		return false;

	int32_t batchSize = 1;
	for (int32_t i = 0; i < pA->rank - 2; i++)
	{
		batchSize *= pA->shape[i];
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput))
		return false;

	int32_t* outShape = (int32_t*)malloc(sizeof(int32_t) * pA->rank);
	if (!outShape)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	for (int32_t i = 0; i < pA->rank - 2; i++)
	{
		outShape[i] = pA->shape[i];
	}
	outShape[pA->rank - 2] = m;
	outShape[pA->rank - 1] = n;

	if (!lySetTensorShape(pOutput, outShape, pA->rank) || !lySetTensorData(pOutput, NULL, getTotalSize(outShape, pA->rank) * sizeof(hip_bfloat16)))
	{
		free(outShape);
		lyDestroyTensor(pOutput);
		return false;
	}
	free(outShape);

	int32_t* aStrides	= (int32_t*)malloc(sizeof(int32_t) * pA->rank);
	int32_t* bStrides	= (int32_t*)malloc(sizeof(int32_t) * pB->rank);
	int32_t* outStrides = (int32_t*)malloc(sizeof(int32_t) * pA->rank);
	if (!aStrides || !bStrides || !outStrides)
	{
		free(aStrides);
		free(bStrides);
		free(outStrides);
		lyDestroyTensor(pOutput);
		return false;
	}

	int32_t *dAShape, *dBShape, *dAStrides, *dBStrides, *dOutStrides;
	hipMalloc(&dAShape, sizeof(int32_t) * pA->rank);
	hipMalloc(&dBShape, sizeof(int32_t) * pB->rank);
	hipMalloc(&dAStrides, sizeof(int32_t) * pA->rank);
	hipMalloc(&dBStrides, sizeof(int32_t) * pB->rank);
	hipMalloc(&dOutStrides, sizeof(int32_t) * pA->rank);

	hipMemcpy(dAShape, pA->shape, sizeof(int32_t) * pA->rank, hipMemcpyHostToDevice);
	hipMemcpy(dBShape, pB->shape, sizeof(int32_t) * pB->rank, hipMemcpyHostToDevice);
	hipMemcpy(dAStrides, aStrides, sizeof(int32_t) * pA->rank, hipMemcpyHostToDevice);
	hipMemcpy(dBStrides, bStrides, sizeof(int32_t) * pB->rank, hipMemcpyHostToDevice);
	hipMemcpy(dOutStrides, outStrides, sizeof(int32_t) * pA->rank, hipMemcpyHostToDevice);

	int32_t totalElements = batchSize * m * n;
	int32_t blockSize	  = 256;
	int32_t gridSize	  = (totalElements + blockSize - 1) / blockSize;

	tensorMatMulKernel<<<gridSize, blockSize>>>(pOutput->data, pA->data, pB->data, dAStrides, dBStrides, dOutStrides, dAShape, dBShape, pA->rank, batchSize, m, n, k);

	hipFree(dAShape);
	hipFree(dBShape);
	hipFree(dAStrides);
	hipFree(dBStrides);
	hipFree(dOutStrides);
	free(aStrides);
	free(bStrides);
	free(outStrides);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	*ppOutput = pOutput;
	return true;
}

bool lyTensorScaleAndAdd(lyTensor** ppOutput, lyTensor* pInput, const lyTensor* pMask, float scale)
{
	if (!ppOutput || !pInput || !pInput->data)
	{
		return false;
	}

	if (pInput->rank != 2)
	{
		return false;
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput))
	{
		return false;
	}

	if (!lySetTensorShape(pOutput, pInput->shape, pInput->rank) || !lySetTensorData(pOutput, NULL, lyTensorGetElementCount(pOutput) * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	dim3 blockSize(16, 16);
	dim3 gridSize((pInput->shape[1] + blockSize.x - 1) / blockSize.x, (pInput->shape[0] + blockSize.y - 1) / blockSize.y);

	tensorScaleAndAddKernel<<<gridSize, blockSize>>>(pOutput->data, pInput->data, pMask ? pMask->data : nullptr, scale, pInput->shape[0], pInput->shape[1]);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	*ppOutput = pOutput;
	return true;
}

bool lyTensorElementwiseMul(lyTensor** ppOutput, const lyTensor* pA, const lyTensor* pB)
{
	if (!ppOutput || !pA || !pB || !pA->data || !pB->data)
	{
		return false;
	}

	if (pA->rank != pB->rank)
	{
		return false;
	}
	for (int i = 0; i < pA->rank; i++)
	{
		if (pA->shape[i] != pB->shape[i])
		{
			return false;
		}
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput))
	{
		return false;
	}

	if (!lySetTensorShape(pOutput, pA->shape, pA->rank) || !lySetTensorData(pOutput, NULL, lyTensorGetElementCount(pOutput) * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	int totalElements = 1;
	for (int i = 0; i < pA->rank; i++)
	{
		totalElements *= pA->shape[i];
	}

	int blockSize = 256;
	int gridSize  = (totalElements + blockSize - 1) / blockSize;

	tensorElementwiseMulKernel<<<gridSize, blockSize>>>(pOutput->data, pA->data, pB->data, totalElements);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	*ppOutput = pOutput;
	return true;
}

bool lyTensorMakeTriangularMask(lyTensor* pTensor)
{
	if (!pTensor || !pTensor->data || pTensor->rank != 2)
	{
		return false;
	}

	int32_t rows = pTensor->shape[0];
	int32_t cols = pTensor->shape[1];

	dim3 blockSize(16, 16);
	dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

	triangularMaskKernel<<<gridSize, blockSize>>>(pTensor->data, rows, cols);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		return false;
	}

	return true;
}

bool lyTensorArgmax(lyTensor** ppOutput, const lyTensor* pInput, int32_t dim)
{
	if (!ppOutput || !pInput || dim < 0 || dim >= pInput->rank)
	{
		return false;
	}

	int32_t* newShape = (int32_t*)malloc(sizeof(int32_t) * (pInput->rank - 1));
	if (!newShape)
	{
		return false;
	}

	int32_t batchSize = 1;
	int32_t j		  = 0;
	for (int32_t i = 0; i < pInput->rank; i++)
	{
		if (i != dim)
		{
			newShape[j++] = pInput->shape[i];
			batchSize *= pInput->shape[i];
		}
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput))
	{
		free(newShape);
		return false;
	}

	if (!lySetTensorShape(pOutput, newShape, pInput->rank - 1) || !lySetTensorData(pOutput, NULL, lyTensorGetElementCount(pOutput) * sizeof(hip_bfloat16)))
	{
		free(newShape);
		lyDestroyTensor(pOutput);
		return false;
	}
	free(newShape);

	int32_t dimSize	  = pInput->shape[dim];
	int32_t blockSize = 256;
	int32_t numBlocks = (batchSize + blockSize - 1) / blockSize;

	tensorArgmaxKernel<<<numBlocks, blockSize>>>(pOutput->data, pInput->data, batchSize, dimSize);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	*ppOutput = pOutput;
	return true;
}

bool lyTensorOuter(lyTensor** ppOutput, const lyTensor* pA, const lyTensor* pB)
{
	if (!ppOutput || !pA || !pB || !pA->data || !pB->data)
	{
		return false;
	}

	if (pA->rank != 1 || pB->rank != 1)
	{
		return false;
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput))
	{
		return false;
	}

	int32_t outputShape[] = {pA->shape[0], pB->shape[0]};
	if (!lySetTensorShape(pOutput, outputShape, 2) || !lySetTensorData(pOutput, NULL, lyTensorGetElementCount(pOutput) * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	dim3 blockSize(16, 16);
	dim3 gridSize((pB->shape[0] + blockSize.x - 1) / blockSize.x, (pA->shape[0] + blockSize.y - 1) / blockSize.y);

	tensorOuterKernel<<<gridSize, blockSize>>>(pOutput->data, pA->data, pB->data, pA->shape[0], pB->shape[0]);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	*ppOutput = pOutput;
	return true;
}

bool lyTensorEmbedding(lyTensor** ppOutput, const lyTensor* pTokens, const lyTensor* pEmbeddings)
{
	if (!ppOutput || !pTokens || !pEmbeddings || pTokens->rank != 1 || pEmbeddings->rank != 2)
	{
		return false;
	}

	int seqLen = pTokens->shape[0];
	int dim	   = pEmbeddings->shape[1];

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput))
	{
		return false;
	}

	int32_t outputShape[] = {seqLen, dim};
	if (!lySetTensorShape(pOutput, outputShape, 2) || !lySetTensorData(pOutput, NULL, seqLen * dim * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	int totalElements = seqLen * dim;
	int blockSize	  = 256;
	int numBlocks	  = (totalElements + blockSize - 1) / blockSize;

	tensorEmbeddingKernel<<<numBlocks, blockSize>>>(pOutput->data, pTokens->data, pEmbeddings->data, seqLen, dim);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	*ppOutput = pOutput;
	return true;
}

bool lyTensorTranspose(lyTensor** ppOutput, const lyTensor* pInput, const int32_t* perm)
{
	if (!ppOutput || !pInput || !pInput->data || !perm || pInput->rank < 2)
	{
		return false;
	}

	int32_t rank		 = pInput->rank;
	int32_t elementCount = lyTensorGetElementCount(pInput);

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput))
	{
		return false;
	}

	int32_t* outputShape = (int32_t*)malloc(rank * sizeof(int32_t));
	if (!outputShape)
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	for (int32_t i = 0; i < rank; i++)
	{
		outputShape[i] = pInput->shape[perm[i]];
	}

	if (!lySetTensorShape(pOutput, outputShape, rank) || !lySetTensorData(pOutput, NULL, elementCount * sizeof(hip_bfloat16)))
	{
		free(outputShape);
		lyDestroyTensor(pOutput);
		return false;
	}
	free(outputShape);

	int32_t* inputStrides  = (int32_t*)malloc(rank * sizeof(int32_t));
	int32_t* outputStrides = (int32_t*)malloc(rank * sizeof(int32_t));
	if (!inputStrides || !outputStrides)
	{
		free(inputStrides);
		free(outputStrides);
		lyDestroyTensor(pOutput);
		return false;
	}

	calculateStrides(inputStrides, pInput->shape, rank);
	calculateStrides(outputStrides, pOutput->shape, rank);

	int32_t *dInputShape, *dInputStrides, *dOutputStrides, *dPerm;
	hipMalloc(&dInputShape, sizeof(int32_t) * rank);
	hipMalloc(&dInputStrides, sizeof(int32_t) * rank);
	hipMalloc(&dOutputStrides, sizeof(int32_t) * rank);
	hipMalloc(&dPerm, sizeof(int32_t) * rank);

	hipMemcpy(dInputShape, pInput->shape, sizeof(int32_t) * rank, hipMemcpyHostToDevice);
	hipMemcpy(dInputStrides, inputStrides, sizeof(int32_t) * rank, hipMemcpyHostToDevice);
	hipMemcpy(dOutputStrides, outputStrides, sizeof(int32_t) * rank, hipMemcpyHostToDevice);
	hipMemcpy(dPerm, perm, sizeof(int32_t) * rank, hipMemcpyHostToDevice);

	int32_t blockSize = 256;
	int32_t gridSize  = (elementCount + blockSize - 1) / blockSize;

	tensorTransposeKernel<<<gridSize, blockSize>>>(pOutput->data, pInput->data, dInputShape, dInputStrides, dOutputStrides, dPerm, rank, elementCount);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		hipFree(dInputShape);
		hipFree(dInputStrides);
		hipFree(dOutputStrides);
		hipFree(dPerm);
		free(inputStrides);
		free(outputStrides);
		lyDestroyTensor(pOutput);
		return false;
	}

	hipFree(dInputShape);
	hipFree(dInputStrides);
	hipFree(dOutputStrides);
	hipFree(dPerm);
	free(inputStrides);
	free(outputStrides);

	*ppOutput = pOutput;
	return true;
}