#include "hip/hip_runtime.h"
#include "lyRotaryPosEmbeddings.h"
#include "lyTensorMath.h"

#include <stdio.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

static bool applyScaling(lyTensor* pFreqs)
{
	if (!pFreqs)
	{
		return false;
	}

	const float scaleFactor		= 8.0f;
	const float lowFreqFactor	= 1.0f;
	const float highFreqFactor	= 4.0f;
	const float oldContextLen	= 8192.0f;	// original llama3 length
	const float lowFreqWavelen	= oldContextLen / lowFreqFactor;
	const float highFreqWavelen = oldContextLen / highFreqFactor;

	int size = pFreqs->shape[0];

	for (int i = 0; i < size; i++)
	{
		float freq;
		if (!lyTensorGetItemAsFloat32(&freq, pFreqs, i))
		{
			return false;
		}

		float wavelen = 2.0f * M_PI / freq;
		float newFreq;

		if (wavelen < highFreqWavelen)
		{
			newFreq = freq;
		}
		else if (wavelen > lowFreqWavelen)
		{
			newFreq = freq / scaleFactor;
		}
		else
		{
			float smooth = (oldContextLen / wavelen - lowFreqFactor) / (highFreqFactor - lowFreqFactor);
			newFreq		 = (1.0f - smooth) * freq / scaleFactor + smooth * freq;
		}

		if (!lyTensorSetItemFromFloat32(pFreqs, i, newFreq))
		{
			return false;
		}
	}

	return true;
}

__global__ void applyRotaryEmbeddingKernel(hip_bfloat16* xqOut, hip_bfloat16* xkOut, const hip_bfloat16* xq, const hip_bfloat16* xk, const hip_bfloat16* freqsCosReal, const hip_bfloat16* freqsSinReal, int batchSize, int headDim)
{
	int idx		  = blockIdx.x * blockDim.x + threadIdx.x;
	int totalSize = batchSize * headDim / 2;

	if (idx >= totalSize)
	{
		return;
	}

	int row = idx / (headDim / 2);
	int col = (idx % (headDim / 2)) * 2;

	hip_bfloat16 xq_real = xq[row * headDim + col];
	hip_bfloat16 xq_imag = xq[row * headDim + col + 1];
	hip_bfloat16 xk_real = xk[row * headDim + col];
	hip_bfloat16 xk_imag = xk[row * headDim + col + 1];

	hip_bfloat16 cos_real = freqsCosReal[col / 2];
	hip_bfloat16 sin_real = freqsSinReal[col / 2];

	hip_bfloat16 xq_out_real = __hsub(__hmul(xq_real, cos_real), __hmul(xq_imag, sin_real));
	hip_bfloat16 xq_out_imag = __hadd(__hmul(xq_real, sin_real), __hmul(xq_imag, cos_real));
	hip_bfloat16 xk_out_real = __hsub(__hmul(xk_real, cos_real), __hmul(xk_imag, sin_real));
	hip_bfloat16 xk_out_imag = __hadd(__hmul(xk_real, sin_real), __hmul(xk_imag, cos_real));

	xqOut[row * headDim + col]	   = xq_out_real;
	xqOut[row * headDim + col + 1] = xq_out_imag;
	xkOut[row * headDim + col]	   = xk_out_real;
	xkOut[row * headDim + col + 1] = xk_out_imag;
}

bool precomputeFreqsCis(lyTensor** ppOut, int32_t dim, int32_t end, float theta)
{
	if (!ppOut || dim <= 0 || end <= 0 || theta <= 0)
	{
		return false;
	}

	lyTensor* freqs;
	int32_t	  freqsShape[] = {dim / 2};
	if (!lyCreateTensor(&freqs))
	{
		return false;
	}
	if (!lySetTensorShape(freqs, freqsShape, 1) || !lySetTensorData(freqs, NULL, (dim / 2) * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(freqs);
		return false;
	}

	float dimFloat = (float)dim;
	for (int i = 0; i < dim / 2; i++)
	{
		float val = (float)(1.0 / pow(theta, (2.0f * i) / dimFloat));
		if (!lyTensorSetItemFromFloat32(freqs, i, val))
		{
			lyDestroyTensor(freqs);
			return false;
		}
	}

	if (!applyScaling(freqs))
	{
		lyDestroyTensor(freqs);
		return false;
	}
	for (int i = 0; i < dim / 2; i++)
	{
		float val;
		lyTensorGetItemAsFloat32(&val, freqs, i);
	}

	lyTensor* t;
	int32_t	  tShape[] = {end};
	if (!lyCreateTensor(&t))
	{
		lyDestroyTensor(freqs);
		return false;
	}
	if (!lySetTensorShape(t, tShape, 1) || !lySetTensorData(t, NULL, end * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(t);
		lyDestroyTensor(freqs);
		return false;
	}

	for (int i = 0; i < end; i++)
	{
		if (!lyTensorSetItemFromFloat32(t, i, (float)i))
		{
			lyDestroyTensor(t);
			lyDestroyTensor(freqs);
			return false;
		}
	}

	lyTensor* freqsOuter;
	if (!lyTensorOuter(&freqsOuter, t, freqs))
	{
		lyDestroyTensor(t);
		lyDestroyTensor(freqs);
		return false;
	}

	for (int i = 0; i < 5; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			float val;
			lyTensorGetItemAsFloat32(&val, freqsOuter, i * dim / 2 + j);
		}
	}

	int32_t	  outShape[] = {end, dim / 2};
	lyTensor* out;
	if (!lyCreateTensor(&out))
	{
		lyDestroyTensor(freqsOuter);
		lyDestroyTensor(t);
		lyDestroyTensor(freqs);
		return false;
	}
	if (!lySetTensorShape(out, outShape, 2) || !lySetTensorData(out, NULL, end * dim * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(out);
		lyDestroyTensor(freqsOuter);
		lyDestroyTensor(t);
		lyDestroyTensor(freqs);
		return false;
	}

	for (int i = 0; i < end; i++)
	{
		for (int j = 0; j < dim / 2; j++)
		{
			float angle;
			if (!lyTensorGetItemAsFloat32(&angle, freqsOuter, i * dim / 2 + j))
			{
				lyDestroyTensor(out);
				lyDestroyTensor(freqsOuter);
				lyDestroyTensor(t);
				lyDestroyTensor(freqs);
				return false;
			}

			float cos_val = cosf(angle);
			float sin_val = sinf(angle);

			if (!lyTensorSetComplexItem(out, i, j, cos_val, sin_val))
			{
				lyDestroyTensor(out);
				lyDestroyTensor(freqsOuter);
				lyDestroyTensor(t);
				lyDestroyTensor(freqs);
				return false;
			}
		}

		hipDeviceSynchronize();
	}

	lyDestroyTensor(freqsOuter);
	lyDestroyTensor(t);
	lyDestroyTensor(freqs);

	*ppOut = out;
	return true;
}

bool lyApplyRotaryEmbedding(lyTensor** ppXQOut, lyTensor** ppXKOut, const lyTensor* pXQ, const lyTensor* pXK, const lyTensor* pFreqsCis)
{
	if (!ppXQOut || !ppXKOut || !pXQ || !pXK || !pFreqsCis)
	{
		return false;
	}

	lyTensor *pXQOut, *pXKOut;
	if (!lyCreateTensor(&pXQOut) || !lyCreateTensor(&pXKOut))
	{
		return false;
	}

	int batchSize	  = pXQ->shape[0];
	int headDim		  = pXQ->shape[1];
	int totalElements = batchSize * headDim / 2;

	if (!lySetTensorShape(pXQOut, pXQ->shape, pXQ->rank) || !lySetTensorData(pXQOut, NULL, totalElements * sizeof(hip_bfloat16)) || !lySetTensorShape(pXKOut, pXK->shape, pXK->rank) || !lySetTensorData(pXKOut, NULL, totalElements * sizeof(hip_bfloat16)))
	{
		lyDestroyTensor(pXQOut);
		lyDestroyTensor(pXKOut);
		return false;
	}

	int blockSize = 256;
	int gridSize  = (totalElements + blockSize - 1) / blockSize;

	applyRotaryEmbeddingKernel<<<gridSize, blockSize>>>(pXQOut->data,
														pXKOut->data,
														pXQ->data,
														pXK->data,
														pFreqsCis->data,	  // Real part (cos)
														pFreqsCis->data + 1,  // Imaginary part (sin)
														batchSize,
														headDim);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		lyDestroyTensor(pXQOut);
		lyDestroyTensor(pXKOut);
		return false;
	}

	*ppXQOut = pXQOut;
	*ppXKOut = pXKOut;
	return true;
}