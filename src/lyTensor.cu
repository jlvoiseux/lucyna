#include "hip/hip_runtime.h"
#include "lyTensor.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

static bool validateIndex(lyTensor* pTensor, int32_t index, size_t elementSize)
{
	if (!pTensor || !pTensor->data)
	{
		return false;
	}

	size_t offset = index * elementSize;
	return offset < pTensor->dataSize;
}

static bool validateComplexAccess(lyTensor* pTensor, int32_t row, int32_t col)
{
	if (!pTensor || !pTensor->data || pTensor->rank != 2)
	{
		return false;
	}

	return row < pTensor->shape[0] && col < pTensor->shape[1];
}

static int32_t calculateIndex(lyTensor* pTensor, const int32_t* pLoc)
{
	int32_t index  = 0;
	int32_t stride = 1;

	for (int32_t i = pTensor->rank - 1; i >= 0; i--)
	{
		if (pLoc[i] >= pTensor->shape[i])
		{
			return -1;
		}
		index += pLoc[i] * stride;
		stride *= pTensor->shape[i];
	}

	return index;
}

__global__ void setItemFromInt32Kernel(hip_bfloat16* data, int32_t index, int32_t value)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		data[index] = __float2bfloat16((float)value);
	}
}

__global__ void setItemFromFloat32Kernel(hip_bfloat16* data, int32_t index, float value)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		data[index] = __float2bfloat16(value);
	}
}

__global__ void setComplexItemKernel(hip_bfloat16* data, int32_t baseIdx, float real, float imag)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		data[baseIdx]	  = __float2bfloat16(real);
		data[baseIdx + 1] = __float2bfloat16(imag);
	}
}

__global__ void getItemKernel(hip_bfloat16* result, const hip_bfloat16* data, int32_t index)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		result[0] = data[index];
	}
}

__global__ void getComplexItemKernel(hip_bfloat16* result, const hip_bfloat16* data, int32_t baseIdx)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		result[0] = data[baseIdx];
		result[1] = data[baseIdx + 1];
	}
}

bool lyCreateTensor(lyTensor** ppTensor, lyMemoryType memoryType)
{
	if (!ppTensor)
	{
		return false;
	}

	lyTensor* pTensor = (lyTensor*)malloc(sizeof(lyTensor));
	if (!pTensor)
	{
		return false;
	}

	memset(pTensor, 0, sizeof(lyTensor));
	pTensor->name		= NULL;
	pTensor->shape		= NULL;
	pTensor->rank		= 0;
	pTensor->data		= NULL;
	pTensor->dataSize	= 0;
	pTensor->memoryType = memoryType;

	*ppTensor = pTensor;
	return true;
}

void lyFreeTensorData(lyTensor* pTensor)
{
	if (!pTensor)
	{
		return;
	}

	if (pTensor->name)
	{
		free(pTensor->name);
		pTensor->name = NULL;
	}

	if (pTensor->shape)
	{
		free(pTensor->shape);
		pTensor->shape = NULL;
	}

	if (pTensor->data)
	{
		if (pTensor->memoryType == LY_MEMORY_GPU)
		{
			hipFree(pTensor->data);
		}
		else
		{
			free(pTensor->data);
		}
	}
	pTensor->data = NULL;
}

void lyDestroyTensor(lyTensor* pTensor)
{
	lyFreeTensorData(pTensor);
	free(pTensor);
}

bool lySetTensorShape(lyTensor* pTensor, const int32_t* pShape, int32_t rank)
{
	if (!pTensor || !pShape || rank <= 0)
	{
		return false;
	}

	int32_t* newShape = (int32_t*)malloc(sizeof(int32_t) * rank);
	if (!newShape)
	{
		return false;
	}

	memcpy(newShape, pShape, sizeof(int32_t) * rank);

	free(pTensor->shape);
	pTensor->shape = newShape;
	pTensor->rank  = rank;

	return true;
}

static bool checkGpuMemory(const char* operation, size_t requestedBytes)
{
	size_t		free, total;
	hipError_t error = hipMemGetInfo(&free, &total);
	if (error != hipSuccess)
	{
		printf("Failed to query GPU memory: %s\n", hipGetErrorString(error));
		return false;
	}

	double freeGb	   = free / (1024.0 * 1024.0 * 1024.0);
	double totalGb	   = total / (1024.0 * 1024.0 * 1024.0);
	double requestedGb = requestedBytes / (1024.0 * 1024.0 * 1024.0);

	printf("GPU Memory Status for %s:\n", operation);
	printf("  Total: %.2f GB\n", totalGb);
	printf("  Free: %.2f GB\n", freeGb);
	printf("  Requested: %.2f GB\n", requestedGb);

	return free >= requestedBytes;
}

bool lySetTensorData(lyTensor* pTensor, const hip_bfloat16* pData, size_t dataSize)
{
	if (!pTensor)
	{
		return false;
	}

	if (pTensor->data)
	{
		if (pTensor->memoryType == LY_MEMORY_GPU)
		{
			hipFree(pTensor->data);
		}
		else
		{
			free(pTensor->data);
		}
		pTensor->data = NULL;
	}

	if (pTensor->memoryType == LY_MEMORY_GPU)
	{
		//		if (!checkGpuMemory("tensor allocation", dataSize))
		//		{
		//			printf("Not enough GPU memory for tensor allocation\n");
		//			return false;
		//		}

		hip_bfloat16* gpuData	 = NULL;
		size_t		 alignedSize = (dataSize + 15) & ~15;  // Align to 16 bytes
		hipError_t	 error		 = hipMalloc(&gpuData, alignedSize);
		if (error != hipSuccess)
		{
			printf("CUDA malloc failed: %s\n", hipGetErrorString(error));
			return false;
		}

		if (pData)
		{
			error = hipMemcpy(gpuData, pData, dataSize, hipMemcpyHostToDevice);
			if (error != hipSuccess)
			{
				printf("CUDA memcpy failed: %s\n", hipGetErrorString(error));
				hipFree(gpuData);
				return false;
			}
		}
		else
		{
			error = hipMemset(gpuData, 0, dataSize);
			if (error != hipSuccess)
			{
				printf("CUDA memset failed: %s\n", hipGetErrorString(error));
				hipFree(gpuData);
				return false;
			}
		}

		pTensor->data = gpuData;
	}
	else
	{
		hip_bfloat16* cpuData = (hip_bfloat16*)malloc(dataSize);
		if (!cpuData)
		{
			return false;
		}

		if (pData)
		{
			memcpy(cpuData, pData, dataSize);
		}
		else
		{
			memset(cpuData, 0, dataSize);
		}

		pTensor->data = cpuData;
	}

	pTensor->dataSize = dataSize;
	return true;
}

bool lySetTensorName(lyTensor* pTensor, const char* name)
{
	if (!pTensor || !name)
	{
		return false;
	}

	char* newName = (char*)malloc(strlen(name) + 1);
	if (!newName)
	{
		return false;
	}

	strcpy(newName, name);
	free(pTensor->name);
	pTensor->name = newName;

	return true;
}

bool lyReshapeTensor(lyTensor* pTensor, const int32_t* pShape, int32_t rank)
{
	if (!pTensor || !pShape || rank <= 0)
	{
		return false;
	}

	int32_t* newShape = (int32_t*)malloc(sizeof(int32_t) * rank);
	if (!newShape)
	{
		return false;
	}

	int32_t oldSize = 1;
	for (int32_t i = 0; i < pTensor->rank; i++)
	{
		oldSize *= pTensor->shape[i];
	}

	int32_t newSize = 1;
	for (int32_t i = 0; i < rank; i++)
	{
		newSize *= pShape[i];
		newShape[i] = pShape[i];
	}

	if (oldSize != newSize)
	{
		free(newShape);
		return false;
	}

	free(pTensor->shape);
	pTensor->shape = newShape;
	pTensor->rank  = rank;

	return true;
}

bool lyTensorSlice(lyTensor** ppOutput, lyTensor* pInput, int32_t startIdx, int32_t endIdx)
{
	if (!ppOutput || !pInput || startIdx < 0 || endIdx <= startIdx || endIdx > pInput->shape[0] || !pInput->data)
	{
		return false;
	}

	lyTensor* pOutput;
	if (!lyCreateTensor(&pOutput, LY_MEMORY_CPU))
	{
		return false;
	}

	int32_t* newShape = (int32_t*)malloc(sizeof(int32_t) * pInput->rank);
	if (!newShape)
	{
		lyDestroyTensor(pOutput);
		return false;
	}
	memcpy(newShape, pInput->shape, sizeof(int32_t) * pInput->rank);
	newShape[0] = endIdx - startIdx;

	if (!lySetTensorShape(pOutput, newShape, pInput->rank))
	{
		free(newShape);
		lyDestroyTensor(pOutput);
		return false;
	}
	free(newShape);

	size_t sliceElements = 1;
	for (int32_t i = 1; i < pInput->rank; i++)
	{
		sliceElements *= pInput->shape[i];
	}
	size_t sliceSize = sliceElements * sizeof(hip_bfloat16);
	size_t offset	 = startIdx * sliceElements;

	if (!lySetTensorData(pOutput, pInput->data + offset, (endIdx - startIdx) * sliceSize))
	{
		lyDestroyTensor(pOutput);
		return false;
	}

	*ppOutput = pOutput;
	return true;
}

bool lyTensorSetItem(lyTensor* pTensor, const int32_t* pLoc, int32_t value)
{
	if (pTensor->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pTensor);
	}

	if (!pTensor || !pLoc)
	{
		return false;
	}

	int32_t index = calculateIndex(pTensor, pLoc);
	if (index < 0)
	{
		return false;
	}

	hipDeviceSynchronize();
	setItemFromInt32Kernel<<<1, 1>>>(pTensor->data, index, value);

	lyTensorMoveToCPU(pTensor);

	return hipGetLastError() == hipSuccess;
}

bool lyTensorGetItem(int32_t* pValue, lyTensor* pTensor, const int32_t* pLoc)
{
	if (!pValue || !pTensor || !pLoc)
	{
		return false;
	}

	int32_t index = calculateIndex(pTensor, pLoc);
	if (index < 0)
	{
		return false;
	}

	float value;
	if (!lyTensorGetItemAsFloat32(&value, pTensor, index))
	{
		return false;
	}

	*pValue = (int32_t)value;
	return true;
}

bool lyTensorGetItemAsFloat32(float* pOut, lyTensor* pTensor, int32_t index)
{
	if (pTensor->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pTensor);
	}

	if (!pOut || !validateIndex(pTensor, index, sizeof(hip_bfloat16)))
	{
		return false;
	}

	hip_bfloat16* temp;
	if (hipMalloc(&temp, sizeof(hip_bfloat16)) != hipSuccess)
	{
		return false;
	}

	hipDeviceSynchronize();
	getItemKernel<<<1, 1>>>(temp, pTensor->data, index);

	hip_bfloat16 hostValue;
	if (hipMemcpy(&hostValue, temp, sizeof(hip_bfloat16), hipMemcpyDeviceToHost) != hipSuccess)
	{
		hipFree(temp);
		return false;
	}

	hipFree(temp);
	*pOut = __bfloat162float(hostValue);

	lyTensorMoveToCPU(pTensor);

	return true;
}

bool lyTensorSetItemFromFloat32(lyTensor* pTensor, int32_t index, float value)
{
	if (pTensor->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pTensor);
	}

	if (!validateIndex(pTensor, index, sizeof(hip_bfloat16)))
	{
		return false;
	}

	hipDeviceSynchronize();
	setItemFromFloat32Kernel<<<1, 1>>>(pTensor->data, index, value);

	lyTensorMoveToCPU(pTensor);

	return hipGetLastError() == hipSuccess;
}

bool lyTensorGetComplexItem(float* pReal, float* pImag, lyTensor* pTensor, int32_t row, int32_t col)
{
	if (pTensor->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pTensor);
	}

	if (!pReal || !pImag || !validateComplexAccess(pTensor, row, col))
	{
		return false;
	}

	int32_t baseIdx = row * pTensor->shape[1] * 2 + col * 2;

	hip_bfloat16* temp;
	if (hipMalloc(&temp, 2 * sizeof(hip_bfloat16)) != hipSuccess)
	{
		return false;
	}

	hipDeviceSynchronize();
	getComplexItemKernel<<<1, 1>>>(temp, pTensor->data, baseIdx);

	hip_bfloat16 hostValues[2];
	if (hipMemcpy(hostValues, temp, 2 * sizeof(hip_bfloat16), hipMemcpyDeviceToHost) != hipSuccess)
	{
		hipFree(temp);
		return false;
	}

	hipFree(temp);
	*pReal = __bfloat162float(hostValues[0]);
	*pImag = __bfloat162float(hostValues[1]);

	lyTensorMoveToCPU(pTensor);

	return true;
}

bool lyTensorSetComplexItem(lyTensor* pTensor, int32_t row, int32_t col, float real, float imag)
{
	if (pTensor->memoryType == LY_MEMORY_CPU)
	{
		lyTensorMoveToGPU(pTensor);
	}

	if (!validateComplexAccess(pTensor, row, col))
	{
		return false;
	}

	int32_t baseIdx = row * pTensor->shape[1] * 2 + col * 2;

	hipDeviceSynchronize();
	setComplexItemKernel<<<1, 1>>>(pTensor->data, baseIdx, real, imag);

	lyTensorMoveToCPU(pTensor);

	return hipGetLastError() == hipSuccess;
}

void lyTensorPrint(lyTensor* pTensor)
{
	if (!pTensor || !pTensor->data)
	{
		printf("Tensor is null or uninitialized.\n");
		return;
	}

	printf("Tensor Metadata:\n");
	printf("Name: %s\n", pTensor->name ? pTensor->name : "Unnamed");
	printf("Rank: %d\n", pTensor->rank);
	printf("Shape: ");
	for (int32_t i = 0; i < pTensor->rank; i++)
	{
		printf("%d%s", pTensor->shape[i], (i < pTensor->rank - 1) ? " x " : "\n");
	}

	if (pTensor->rank > 3)
	{
		printf("Error: Printing tensors with rank > 3 is not supported.\n");
		return;
	}

	size_t elements = 1;
	for (int32_t i = 0; i < pTensor->rank; i++)
	{
		elements *= pTensor->shape[i];
	}

	hip_bfloat16* hostData = (hip_bfloat16*)malloc(pTensor->dataSize);
	if (!hostData)
	{
		printf("Error: Failed to allocate memory for tensor data.\n");
		return;
	}

	if (hipMemcpy(hostData, pTensor->data, pTensor->dataSize, hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("Error: Failed to copy tensor data from device to host.\n");
		free(hostData);
		return;
	}

	printf("Tensor Data:\n");
	if (pTensor->rank == 1)
	{
		for (int32_t i = 0; i < pTensor->shape[0]; i++)
		{
			printf("%f ", __bfloat162float(hostData[i]));
		}
		printf("\n");
	}
	else if (pTensor->rank == 2)
	{
		for (int32_t i = 0; i < pTensor->shape[0]; i++)
		{
			for (int32_t j = 0; j < pTensor->shape[1]; j++)
			{
				printf("%f ", __bfloat162float(hostData[i * pTensor->shape[1] + j]));
			}
			printf("\n");
		}
	}
	else if (pTensor->rank == 3)
	{
		for (int32_t i = 0; i < pTensor->shape[0]; i++)
		{
			printf("Slice %d:\n", i);
			for (int32_t j = 0; j < pTensor->shape[1]; j++)
			{
				for (int32_t k = 0; k < pTensor->shape[2]; k++)
				{
					printf("%f ", __bfloat162float(hostData[i * pTensor->shape[1] * pTensor->shape[2] + j * pTensor->shape[2] + k]));
				}
				printf("\n");
			}
			printf("\n");
		}
	}

	free(hostData);
}

bool lyTensorMoveToGPU(lyTensor* pTensor)
{
	hipDeviceSynchronize();
	if (!pTensor || pTensor->memoryType != LY_MEMORY_CPU || !pTensor->data)
	{
		return false;
	}

	//	if (!checkGpuMemory("tensor allocation", pTensor->dataSize))
	//	{
	//		printf("Not enough GPU memory for tensor allocation\n");
	//		return false;
	//	}

	hip_bfloat16* gpuData = NULL;
	hipError_t	 error	 = hipMalloc(&gpuData, pTensor->dataSize);
	if (error != hipSuccess)
	{
		return false;
	}

	error = hipMemcpy(gpuData, pTensor->data, pTensor->dataSize, hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		hipFree(gpuData);
		return false;
	}

	free(pTensor->data);
	pTensor->data		= gpuData;
	pTensor->memoryType = LY_MEMORY_GPU;
	hipDeviceSynchronize();
	return true;
}

bool lyTensorMoveToCPU(lyTensor* pTensor)
{
	hipDeviceSynchronize();
	if (!pTensor || pTensor->memoryType != LY_MEMORY_GPU || !pTensor->data)
	{
		return false;
	}

	hip_bfloat16* cpuData = (hip_bfloat16*)malloc(pTensor->dataSize);
	if (!cpuData)
	{
		return false;
	}

	hipError_t error = hipMemcpy(cpuData, pTensor->data, pTensor->dataSize, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		free(cpuData);
		return false;
	}

	hipFree(pTensor->data);
	pTensor->data		= cpuData;
	pTensor->memoryType = LY_MEMORY_CPU;

	hipDeviceSynchronize();
	return true;
}