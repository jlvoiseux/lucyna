#include "hip/hip_runtime.h"
#include "lyTensor.h"
#include "lyTensorMath.h"
#include "unity.h"

static lyTensor* pTensorA = NULL;
static lyTensor* pTensorB = NULL;

void setUp(void) {}

void tearDown(void)
{
	if (pTensorA)
	{
		lyTensorDestroy(pTensorA);
		pTensorA = NULL;
	}
	if (pTensorB)
	{
		lyTensorDestroy(pTensorB);
		pTensorB = NULL;
	}
}

void test_TensorScaleAndAdd2D(void)
{
	int32_t shape[] = {2, 3};

	hip_bfloat16 dataA[6];
	for (int i = 0; i < 6; i++)
	{
		dataA[i] = __float2bfloat16_rz((float)(i + 1));
	}

	hip_bfloat16 dataB[6];
	for (int i = 0; i < 6; i++)
	{
		dataB[i] = __float2bfloat16_rz((float)(i + 1) * 0.5f);
	}

	lyTensorCreate(&pTensorA, shape, 2, dataA, NULL);
	lyTensorCreate(&pTensorB, shape, 2, dataB, NULL);

	lyTensor* pOutput;
	float	  alpha = 2.0f;
	float	  beta	= -1.0f;
	lyTensorScaleAndAdd(&pOutput, pTensorA, pTensorB, alpha, beta);

	float expected[] = {1.5f, 3.0f, 4.5f, 6.0f, 7.5f, 9.0f};
	for (int i = 0; i < 6; i++)
	{
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected[i], __bfloat162float(pOutput->data[i]));
	}

	lyTensorDestroy(pOutput);
}

void test_TensorScaleAndAdd3D(void)
{
	int32_t		 shape[]  = {2, 2, 2};
	size_t		 elements = 8;
	hip_bfloat16* dataA	  = (hip_bfloat16*)malloc(elements * sizeof(hip_bfloat16));
	hip_bfloat16* dataB	  = (hip_bfloat16*)malloc(elements * sizeof(hip_bfloat16));

	for (size_t i = 0; i < elements; i++)
	{
		dataA[i] = __float2bfloat16_rz((float)(i + 1));
		dataB[i] = __float2bfloat16_rz((float)(i + 1) * 0.1f);
	}

	lyTensorCreate(&pTensorA, shape, 3, dataA, NULL);
	lyTensorCreate(&pTensorB, shape, 3, dataB, NULL);

	lyTensor* pOutput;
	float	  alpha = 0.5f;
	float	  beta	= 2.0f;
	lyTensorScaleAndAdd(&pOutput, pTensorA, pTensorB, alpha, beta);

	for (size_t i = 0; i < elements; i++)
	{
		float expected = 0.5f * (float)(i + 1) + 2.0f * ((float)(i + 1) * 0.1f);
		float actual   = __bfloat162float(pOutput->data[i]);
		TEST_ASSERT_FLOAT_WITHIN(0.05f, expected, actual);
	}

	free(dataA);
	free(dataB);
	lyTensorDestroy(pOutput);
}

void test_TensorScaleAndAddInvalidShapes(void)
{
	int32_t shapeA[] = {2, 3};
	int32_t shapeB[] = {2, 3, 2};
	lyTensorCreate(&pTensorA, shapeA, 2, NULL, NULL);
	lyTensorCreate(&pTensorB, shapeB, 3, NULL, NULL);

	lyTensor* pOutput;
	lyTensorScaleAndAdd(&pOutput, pTensorA, pTensorB, 1.0f, 1.0f);

	// Test tensors with same rank but different dimensions
	int32_t shapeC[] = {2, 4};
	lyTensorCreate(&pTensorB, shapeC, 2, NULL, NULL);
	lyTensorScaleAndAdd(&pOutput, pTensorA, pTensorB, 1.0f, 1.0f);
}

void test_TensorScaleAndAddRank1Invalid(void)
{
	int32_t shape[] = {3};
	lyTensorCreate(&pTensorA, shape, 1, NULL, NULL);
	lyTensorCreate(&pTensorB, shape, 1, NULL, NULL);
	lyTensor* pOutput;
	lyTensorScaleAndAdd(&pOutput, pTensorA, pTensorB, 1.0f, 1.0f);
}

void test_TensorScaleAndAddBroadcast(void)
{
	// Create tensor A with shape (2, 3, 4)
	int32_t shapeA[] = {2, 3, 4};

	// Create tensor B with shape (3, 4)
	int32_t shapeB[] = {3, 4};

	// Initialize tensor A with values 1,2,3,...,24
	hip_bfloat16* dataA = (hip_bfloat16*)malloc(24 * sizeof(hip_bfloat16));
	for (int i = 0; i < 24; i++)
	{
		dataA[i] = __float2bfloat16_rz((float)(i + 1));
	}

	// Initialize tensor B with values 0.1,0.2,0.3,...,1.2
	hip_bfloat16* dataB = (hip_bfloat16*)malloc(12 * sizeof(hip_bfloat16));
	for (int i = 0; i < 12; i++)
	{
		dataB[i] = __float2bfloat16_rz((float)(i + 1) * 0.1f);
	}

	lyTensorCreate(&pTensorA, shapeA, 3, dataA, NULL);
	lyTensorCreate(&pTensorB, shapeB, 2, dataB, NULL);

	lyTensor* pOutput;
	float	  alpha = 2.0f;
	float	  beta	= -1.0f;
	lyTensorScaleAndAdd(&pOutput, pTensorA, pTensorB, alpha, beta);

	TEST_ASSERT_FLOAT_WITHIN(0.05f, 1.9f, __bfloat162float(pOutput->data[0]));
	TEST_ASSERT_FLOAT_WITHIN(0.05f, 3.8f, __bfloat162float(pOutput->data[1]));
	TEST_ASSERT_FLOAT_WITHIN(0.05f, 5.7f, __bfloat162float(pOutput->data[2]));

	free(dataA);
	free(dataB);
	lyTensorDestroy(pOutput);
}

void test_TensorScaleOnly(void)
{
	int32_t shape[] = {2, 3};

	hip_bfloat16 dataA[6];
	for (int i = 0; i < 6; i++)
	{
		dataA[i] = (float)(i + 1);
	}

	lyTensorCreate(&pTensorA, shape, 2, dataA, NULL);

	lyTensor* pOutput;
	float	  alpha = 2.0f;
	lyTensorScaleAndAdd(&pOutput, pTensorA, NULL, alpha, 0.0f);	 // beta unused when pB is NULL

	// Check that each element was properly scaled
	for (int i = 0; i < 6; i++)
	{
		float expected = (float)(i + 1) * alpha;
		float actual   = __bfloat162float(pOutput->data[i]);
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected, actual);
	}

	lyTensorDestroy(pOutput);
}

void test_MatMul2D(void)
{
	int32_t shapeA[] = {2, 3};
	int32_t shapeB[] = {3, 2};

	hip_bfloat16 dataA[6];
	for (int i = 0; i < 6; i++)
	{
		dataA[i] = __float2bfloat16_rz((float)(i + 1));
	}

	hip_bfloat16 dataB[6];
	for (int i = 0; i < 6; i++)
	{
		dataB[i] = __float2bfloat16_rz((float)(i + 1));
	}

	lyTensorCreate(&pTensorA, shapeA, 2, dataA, NULL);
	lyTensorCreate(&pTensorB, shapeB, 2, dataB, NULL);

	lyTensor* pOutput;
	lyTensorMatMul(&pOutput, pTensorA, pTensorB);

	TEST_ASSERT_EQUAL_INT32(2, pOutput->rank);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[0]);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[1]);

	float expected[] = {22.0f, 28.0f, 49.0f, 64.0f};
	for (int i = 0; i < 4; i++)
	{
		TEST_ASSERT_FLOAT_WITHIN(0.1f, expected[i], __bfloat162float(pOutput->data[i]));
	}

	lyTensorDestroy(pOutput);
}

void test_MatMul3D(void)
{
	// Create small test case with same structure as attention
	// [2, 3, 4] x [2, 4, 3] -> [2, 3, 3]
	int32_t shapeA[] = {2, 3, 4};
	int32_t shapeB[] = {2, 4, 3};

	hip_bfloat16 dataA[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24};
	hip_bfloat16 dataB[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24};

	lyTensor *pA, *pB;
	lyTensorCreate(&pA, shapeA, 3, dataA, NULL);
	lyTensorCreate(&pB, shapeB, 3, dataB, NULL);

	lyTensor* pOutput;
	lyTensorMatMul(&pOutput, pA, pB);

	TEST_ASSERT_EQUAL_INT32(3, pOutput->rank);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[0]);
	TEST_ASSERT_EQUAL_INT32(3, pOutput->shape[1]);
	TEST_ASSERT_EQUAL_INT32(3, pOutput->shape[2]);

	float expected_first[] = {70, 80, 90, 158, 184, 210, 246, 288, 330};

	for (int i = 0; i < 9; i++)
	{
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected_first[i], __bfloat162float(pOutput->data[i]));
	}

	lyTensorDestroy(pA);
	lyTensorDestroy(pB);
	lyTensorDestroy(pOutput);
}

void test_MatMulInvalidShapes(void)
{
	int32_t shapeA[] = {2, 3};
	int32_t shapeB[] = {2, 2};
	lyTensorCreate(&pTensorA, shapeA, 2, NULL, NULL);
	lyTensorCreate(&pTensorB, shapeB, 2, NULL, NULL);

	lyTensor* pOutput;
	lyTensorMatMul(&pOutput, pTensorA, pTensorB);
}

void test_MatMulDifferentRanks(void)
{
	int32_t shapeA[] = {2, 2, 3};
	int32_t shapeB[] = {3, 2};	// Different rank
	lyTensorCreate(&pTensorA, shapeA, 3, NULL, NULL);
	lyTensorCreate(&pTensorB, shapeB, 2, NULL, NULL);

	lyTensor* pOutput;
	lyTensorMatMul(&pOutput, pTensorA, pTensorB);
}

void test_MatMul4D(void)
{
	int32_t shapeA[] = {2, 2, 2, 3};
	int32_t shapeB[] = {2, 2, 3, 2};

	int32_t		 sizeA = 2 * 2 * 2 * 3;
	int32_t		 sizeB = 2 * 2 * 3 * 2;
	hip_bfloat16* dataA = (hip_bfloat16*)malloc(sizeA * sizeof(hip_bfloat16));
	hip_bfloat16* dataB = (hip_bfloat16*)malloc(sizeB * sizeof(hip_bfloat16));

	for (int i = 0; i < sizeA; i++)
	{
		dataA[i] = __float2bfloat16_rz((float)(i + 1));
	}
	for (int i = 0; i < sizeB; i++)
	{
		dataB[i] = __float2bfloat16_rz((float)(i + 1));
	}

	lyTensorCreate(&pTensorA, shapeA, 4, dataA, NULL);
	lyTensorCreate(&pTensorB, shapeB, 4, dataB, NULL);

	lyTensor* pOutput;
	lyTensorMatMul(&pOutput, pTensorA, pTensorB);

	TEST_ASSERT_EQUAL_INT32(4, pOutput->rank);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[0]);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[1]);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[2]);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[3]);

	free(dataA);
	free(dataB);
	lyTensorDestroy(pOutput);
}

void test_TensorElementwiseMulBasic(void)
{
	int32_t shape[] = {2, 2};

	hip_bfloat16 dataA[] = {1.0f, 2.0f, 3.0f, 4.0f};
	hip_bfloat16 dataB[] = {2.0f, 2.0f, 2.0f, 2.0f};

	lyTensor *pA, *pB;
	lyTensorCreate(&pA, shape, 2, dataA, NULL);
	lyTensorCreate(&pB, shape, 2, dataB, NULL);

	lyTensor* pOutput;
	lyTensorElementwiseMul(&pOutput, pA, pB);

	float expected[] = {2.0f, 4.0f, 6.0f, 8.0f};
	for (int i = 0; i < 4; i++)
	{
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected[i], __bfloat162float(pOutput->data[i]));
	}

	lyTensorDestroy(pA);
	lyTensorDestroy(pB);
	lyTensorDestroy(pOutput);
}

void test_TensorElementwiseMulBroadcast(void)
{
	int32_t shapeA[] = {2, 3, 4};
	int32_t shapeB[] = {4};	 // Broadcasting 4 across 2,3,4

	hip_bfloat16* dataA = (hip_bfloat16*)malloc(24 * sizeof(hip_bfloat16));
	hip_bfloat16* dataB = (hip_bfloat16*)malloc(4 * sizeof(hip_bfloat16));

	for (int i = 0; i < 24; i++)
	{
		dataA[i] = __float2bfloat16((float)(i + 1));
	}
	for (int i = 0; i < 4; i++)
	{
		dataB[i] = __float2bfloat16(2.0f);
	}

	lyTensor *pA, *pB;
	lyTensorCreate(&pA, shapeA, 3, dataA, NULL);
	lyTensorCreate(&pB, shapeB, 1, dataB, NULL);

	lyTensor* pOutput;
	lyTensorElementwiseMul(&pOutput, pA, pB);

	TEST_ASSERT_EQUAL_INT32(3, pOutput->rank);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[0]);
	TEST_ASSERT_EQUAL_INT32(3, pOutput->shape[1]);
	TEST_ASSERT_EQUAL_INT32(4, pOutput->shape[2]);

	// Check first few values
	for (int i = 0; i < 4; i++)
	{
		float expected = (float)(i + 1) * 2.0f;
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected, __bfloat162float(pOutput->data[i]));
	}

	free(dataA);
	free(dataB);
	lyTensorDestroy(pA);
	lyTensorDestroy(pB);
	lyTensorDestroy(pOutput);
}

void test_TensorElementwiseMulBroadcast2D(void)
{
	int32_t shapeA[] = {2, 4};
	int32_t shapeB[] = {4};

	hip_bfloat16 dataA[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f};
	hip_bfloat16 dataB[] = {2.0f, 3.0f, 4.0f, 5.0f};

	lyTensor *pA, *pB;
	lyTensorCreate(&pA, shapeA, 2, dataA, NULL);
	lyTensorCreate(&pB, shapeB, 1, dataB, NULL);

	lyTensor* pOutput;
	lyTensorElementwiseMul(&pOutput, pA, pB);

	float expected[] = {2.0f, 6.0f, 12.0f, 20.0f, 10.0f, 18.0f, 28.0f, 40.0f};
	for (int i = 0; i < 8; i++)
	{
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected[i], __bfloat162float(pOutput->data[i]));
	}

	lyTensorDestroy(pA);
	lyTensorDestroy(pB);
	lyTensorDestroy(pOutput);
}

void test_TensorMakeTriangularMask(void)
{
	int32_t shape[] = {3, 3};
	lyTensorCreate(&pTensorA, shape, 2, NULL, NULL);
	lyTensorMakeTriangularMask(pTensorA);

	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 3; j++)
		{
			float expected = j <= i ? 0.0f : -INFINITY;
			float actual   = __bfloat162float(pTensorA->data[i * 3 + j]);
			if (expected == -INFINITY)
			{
				TEST_ASSERT_EQUAL_FLOAT(expected, actual);
			}
			else
			{
				TEST_ASSERT_FLOAT_WITHIN(0.01f, expected, actual);
			}
		}
	}
}

void test_TensorArgmax(void)
{
	int32_t		shape[] = {1, 6};
	hip_bfloat16 data[]	= {1.0f, 3.0f, 2.0f, 0.0f, 5.0f, 4.0f};
	lyTensorCreate(&pTensorA, shape, 2, data, NULL);

	int32_t pOutput;
	lyTensorArgmax(&pOutput, pTensorA);

	TEST_ASSERT_EQUAL_INT32(4, pOutput);
}

void test_TensorSoftmax(void)
{
	// Test 2D tensor softmax on last dim
	int32_t		shape[] = {2, 3};
	hip_bfloat16 data[]	= {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f};
	lyTensorCreate(&pTensorA, shape, 2, data, NULL);

	lyTensor* pOutput;
	lyTensorSoftmax(&pOutput, pTensorA);

	float sum1 = 0.0f;
	for (int i = 0; i < 3; i++)
	{
		sum1 += __bfloat162float(pOutput->data[i]);
	}
	TEST_ASSERT_FLOAT_WITHIN(0.01f, 1.0f, sum1);

	float sum2 = 0.0f;
	for (int i = 3; i < 6; i++)
	{
		sum2 += __bfloat162float(pOutput->data[i]);
	}
	TEST_ASSERT_FLOAT_WITHIN(0.01f, 1.0f, sum2);

	float expected1[] = {0.0900f, 0.2447f, 0.6652f};
	for (int i = 0; i < 3; i++)
	{
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected1[i], __bfloat162float(pOutput->data[i]));
	}

	lyTensorDestroy(pOutput);

	// Test 3D tensor softmax on last dim
	int32_t		shape3d[] = {2, 2, 3};	// Changed from {2,2,2} to {2,2,3}
	hip_bfloat16 data3d[]  = {
		 1.0f,
		 2.0f,
		 3.0f,	// First batch, first row
		 4.0f,
		 5.0f,
		 6.0f,	// First batch, second row
		 7.0f,
		 8.0f,
		 9.0f,	// Second batch, first row
		 10.0f,
		 11.0f,
		 12.0f	// Second batch, second row
	 };
	lyTensorCreate(&pTensorA, shape3d, 3, data3d, NULL);

	lyTensorSoftmax(&pOutput, pTensorA);

	TEST_ASSERT_EQUAL_INT32(3, pOutput->rank);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[0]);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[1]);
	TEST_ASSERT_EQUAL_INT32(3, pOutput->shape[2]);

	// Check that each row sums to 1
	for (int i = 0; i < 2; i++)
	{
		for (int j = 0; j < 2; j++)
		{
			float sum = 0.0f;
			for (int k = 0; k < 3; k++)
			{
				sum += __bfloat162float(pOutput->data[i * 6 + j * 3 + k]);
			}
			TEST_ASSERT_FLOAT_WITHIN(0.01f, 1.0f, sum);
		}
	}

	lyTensorDestroy(pOutput);
}

void test_TensorOuter(void)
{
	int32_t shapeA[] = {2};
	int32_t shapeB[] = {3};

	hip_bfloat16 dataA[] = {1.0f, 2.0f};
	hip_bfloat16 dataB[] = {3.0f, 4.0f, 5.0f};

	lyTensorCreate(&pTensorA, shapeA, 1, dataA, NULL);
	lyTensorCreate(&pTensorB, shapeB, 1, dataB, NULL);

	lyTensor* pOutput;
	lyTensorOuter(&pOutput, pTensorA, pTensorB);

	TEST_ASSERT_EQUAL_INT32(2, pOutput->rank);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[0]);
	TEST_ASSERT_EQUAL_INT32(3, pOutput->shape[1]);

	float expected[6] = {3.0f, 4.0f, 5.0f, 6.0f, 8.0f, 10.0f};
	for (int i = 0; i < 6; i++)
	{
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected[i], __bfloat162float(pOutput->data[i]));
	}

	lyTensorDestroy(pOutput);
}

void test_TensorEmbedding(void)
{
	// Create embedding matrix [4, 2]
	int32_t embeddingShape[] = {4, 2};

	// Set up token IDs: [1, 0, 2]
	int32_t tokenData[3];
	tokenData[0] = 1;
	tokenData[1] = 0;
	tokenData[2] = 2;

	// Set up embedding matrix:
	// [[0.0, 0.1],
	//  [1.0, 1.1],
	//  [2.0, 2.1],
	//  [3.0, 3.1]]
	hip_bfloat16 embeddingData[8];
	for (int i = 0; i < 4; i++)
	{
		embeddingData[i * 2]	 = i;
		embeddingData[i * 2 + 1] = i + 0.1f;
	}

	lyTensorCreate(&pTensorB, embeddingShape, 2, embeddingData, NULL);

	lyTensor* pOutput;
	lyTensorEmbedding(&pOutput, tokenData, 3, pTensorB);

	TEST_ASSERT_EQUAL_INT32(2, pOutput->rank);
	TEST_ASSERT_EQUAL_INT32(3, pOutput->shape[0]);	// sequence length
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[1]);	// embedding dimension

	// Expected sequence:
	// Token 1 -> [1.0, 1.1]
	// Token 0 -> [0.0, 0.1]
	// Token 2 -> [2.0, 2.1]
	float expected[6] = {1.0f, 1.1f, 0.0f, 0.1f, 2.0f, 2.1f};
	for (int i = 0; i < 6; i++)
	{
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected[i], __bfloat162float(pOutput->data[i]));
	}

	lyTensorDestroy(pOutput);
}

void test_TensorTranspose(void)
{
	int32_t		 shape[] = {2, 3};
	hip_bfloat16* data	 = (hip_bfloat16*)malloc(6 * sizeof(hip_bfloat16));
	for (int i = 0; i < 6; i++)
	{
		data[i] = (float)i;
	}
	lyTensorCreate(&pTensorA, shape, 2, data, NULL);
	free(data);

	lyTensor* pOutput;
	int32_t	  perm[] = {1, 0};

	lyTensorTranspose(&pOutput, pTensorA, perm);

	TEST_ASSERT_EQUAL_INT32(2, pOutput->rank);
	TEST_ASSERT_EQUAL_INT32(3, pOutput->shape[0]);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[1]);

	// Expected after transpose:
	// 0 3
	// 1 4
	// 2 5
	float expected[] = {0.0f, 3.0f, 1.0f, 4.0f, 2.0f, 5.0f};
	for (int i = 0; i < 6; i++)
	{
		TEST_ASSERT_FLOAT_WITHIN(0.01f, expected[i], pOutput->data[i]);
	}

	lyTensorDestroy(pOutput);
}

void test_TensorTranspose3D(void)
{
	int32_t		 shape[] = {2, 3, 4};
	hip_bfloat16* data	 = (hip_bfloat16*)malloc(24 * sizeof(hip_bfloat16));
	for (int i = 0; i < 24; i++)
	{
		data[i] = (float)i;
	}
	lyTensorCreate(&pTensorA, shape, 3, data, NULL);
	free(data);

	lyTensor* pOutput;
	int32_t	  perm[] = {0, 2, 1};
	lyTensorTranspose(&pOutput, pTensorA, perm);

	TEST_ASSERT_EQUAL_INT32(3, pOutput->rank);
	TEST_ASSERT_EQUAL_INT32(2, pOutput->shape[0]);
	TEST_ASSERT_EQUAL_INT32(4, pOutput->shape[1]);
	TEST_ASSERT_EQUAL_INT32(3, pOutput->shape[2]);

	TEST_ASSERT_FLOAT_WITHIN(0.01f, 0.0f, pOutput->data[0]);
	TEST_ASSERT_FLOAT_WITHIN(0.01f, 4.0f, pOutput->data[1]);
	TEST_ASSERT_FLOAT_WITHIN(0.01f, 8.0f, pOutput->data[2]);

	lyTensorDestroy(pOutput);
}

void test_TensorTranspose3DLarge(void)
{
	int32_t shape[]	 = {32, 32, 128};
	size_t	elements = 32 * 32 * 128;

	hip_bfloat16* data = (hip_bfloat16*)malloc(elements * sizeof(hip_bfloat16));
	TEST_ASSERT_NOT_NULL(data);

	for (size_t i = 0; i < elements; i++)
	{
		data[i] = (float)(i + 1);
	}

	lyTensorCreate(&pTensorA, shape, 3, data, NULL);

	int32_t	  perm[] = {1, 0, 2};
	lyTensor* pTransposed;
	lyTensorTranspose(&pTransposed, pTensorA, perm);

	TEST_ASSERT_EQUAL_INT32(3, pTransposed->rank);
	TEST_ASSERT_EQUAL_INT32(32, pTransposed->shape[0]);
	TEST_ASSERT_EQUAL_INT32(32, pTransposed->shape[1]);
	TEST_ASSERT_EQUAL_INT32(128, pTransposed->shape[2]);

	for (int i = 0; i < 32; i++)
	{
		for (int j = 0; j < 32; j++)
		{
			for (int k = 0; k < 128; k++)
			{
				size_t originalIndex   = i * 32 * 128 + j * 128 + k;
				size_t transposedIndex = j * 32 * 128 + i * 128 + k;
				TEST_ASSERT_FLOAT_WITHIN(0.01f, data[originalIndex], pTransposed->data[transposedIndex]);
			}
		}
	}

	free(data);
	lyTensorDestroy(pTransposed);
}

int main(void)
{
	UNITY_BEGIN();
	RUN_TEST(test_TensorScaleAndAdd2D);
	RUN_TEST(test_TensorScaleAndAdd3D);
	RUN_TEST(test_TensorScaleAndAddInvalidShapes);
	RUN_TEST(test_TensorScaleAndAddRank1Invalid);
	RUN_TEST(test_TensorScaleAndAddBroadcast);
	RUN_TEST(test_TensorScaleOnly);
	RUN_TEST(test_MatMul2D);
	RUN_TEST(test_MatMul3D);
	RUN_TEST(test_MatMulInvalidShapes);
	RUN_TEST(test_MatMulDifferentRanks);
	RUN_TEST(test_MatMul4D);
	RUN_TEST(test_TensorElementwiseMulBasic);
	RUN_TEST(test_TensorElementwiseMulBroadcast);
	RUN_TEST(test_TensorElementwiseMulBroadcast2D);
	RUN_TEST(test_TensorMakeTriangularMask);
	RUN_TEST(test_TensorArgmax);
	RUN_TEST(test_TensorSoftmax);
	RUN_TEST(test_TensorOuter);
	RUN_TEST(test_TensorEmbedding);
	RUN_TEST(test_TensorTranspose);
	RUN_TEST(test_TensorTranspose3D);
	RUN_TEST(test_TensorTranspose3DLarge);
	return UNITY_END();
}